#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>


float cpuPICalculate(long trials) {
	float x, y;
	long pointsInCircle = 0;
	for (long i = 0; i < trials; i++) {
		x = rand() / (float)RAND_MAX;
		y = rand() / (float)RAND_MAX;
		pointsInCircle += (x*x + y * y <= 1.0f);
	}
	return 4.0f * pointsInCircle / trials;
}

int main(int argc, char *argv[]) {
	clock_t start, stop;

	start = clock();
	float cpuPI = cpuPICalculate(100000000);
	stop = clock();
	printf("CPU PI= %f\n", cpuPI);
	printf("CPU PI calculate time %f s.\n", (stop - start) / (float)CLOCKS_PER_SEC);

	return 0;
}